#include "cuSZp_entry_1D_f64.h"
#include "cuSZp_kernels_1D_f64.h"

/** ************************************************************************
 * @brief cuSZp end-to-end compression API for device pointers
 *        Compression is executed in GPU.
 *        Original data is stored as device pointers (in GPU).
 *        Compressed data is stored back as device pointers (in GPU).
 * 
 * @param   d_oriData       original data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           original data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void cuSZp_compress_1D_plain_f64(double* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = tblock_size;
    int gsize = (nbEle + bsize * thread_chunk - 1) / (bsize * thread_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    cuSZp_compress_kernel_1D_plain_f64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-1, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+tblock_size*thread_chunk-1)/(tblock_size*thread_chunk)*(tblock_size*thread_chunk)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

 /** ************************************************************************
 * @brief cuSZp end-to-end decompression API for device pointers
 *        Decompression is executed in GPU.
 *        Compressed data is stored as device pointers (in GPU).
 *        Reconstructed data is stored as device pointers (in GPU).
 *        P.S. Reconstructed data and original data have the same shape.
 * 
 * @param   d_decData       reconstructed data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           reconstructed data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void cuSZp_decompress_1D_plain_f64(double* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = tblock_size;
    int gsize = (nbEle + bsize * thread_chunk - 1) / (bsize * thread_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU decompression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    cuSZp_decompress_kernel_1D_plain_f64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);
    
    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

/** ************************************************************************
 * @brief cuSZp end-to-end compression API for device pointers
 *        Compression is executed in GPU.
 *        Original data is stored as device pointers (in GPU).
 *        Compressed data is stored back as device pointers (in GPU).
 * 
 * @param   d_oriData       original data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           original data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void cuSZp_compress_1D_outlier_f64(double* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = tblock_size;
    int gsize = (nbEle + bsize * thread_chunk - 1) / (bsize * thread_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    cuSZp_compress_kernel_1D_outlier_f64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-1, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+tblock_size*thread_chunk-1)/(tblock_size*thread_chunk)*(tblock_size*thread_chunk)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

 /** ************************************************************************
 * @brief cuSZp end-to-end decompression API for device pointers
 *        Decompression is executed in GPU.
 *        Compressed data is stored as device pointers (in GPU).
 *        Reconstructed data is stored as device pointers (in GPU).
 *        P.S. Reconstructed data and original data have the same shape.
 * 
 * @param   d_decData       reconstructed data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           reconstructed data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void cuSZp_decompress_1D_outlier_f64(double* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = tblock_size;
    int gsize = (nbEle + bsize * thread_chunk - 1) / (bsize * thread_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU decompression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    cuSZp_decompress_kernel_1D_outlier_f64<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);
    
    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}